#include "hip/hip_runtime.h"
#include "kernel.h"
#include<stdio.h>
#include <stdlib.h> 
#include "hip/hip_runtime.h"
#include ""
#define TPB 32

__global__ void scale_Kernel(float *d_inp, int len, float *d_out, float scale)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<len)
		d_out[i] = d_inp[i] * scale;

}
__global__ void vector_AddKernel(float *d_inp1, int len1, float *d_inp2, int len2, float *d_out)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<len1)
		d_out[i] = d_inp1[i] + d_inp2[i];



}

__global__ void component_ProdKernel(float *d_inp1, int len1, float *d_inp2, int len2, float *d_out)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<len1)
		d_out[i] = d_inp1[i] * d_inp2[i];


}


void scale(float *inp, int len, float *out, float scale)
{
	float *d_inp = 0;
	float *d_out = 0;

	hipMalloc(&d_inp, len*sizeof(float));
	hipMalloc(&d_out, len*sizeof(float));

	hipMemcpy(d_inp, inp, len*sizeof(float), hipMemcpyHostToDevice);
	scale_Kernel << <(len / TPB), TPB >> >(d_inp, len, d_out, scale);

	hipMemcpy(out, d_out, len*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_inp);
	hipFree(d_out);
}

void vector_Add(float *inp1, int len1, float *inp2, int len2, float *out)
{
		float *d_inp1 = 0;
		float *d_inp2 = 0;
		float *d_out = 0;

		hipMalloc(&d_inp1, len1*sizeof(float));
		hipMalloc(&d_inp2, len1*sizeof(float));
		hipMalloc(&d_out, len1*sizeof(float));

		hipMemcpy(d_inp1, inp1, len1*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_inp2, inp2, len1*sizeof(float), hipMemcpyHostToDevice);

		vector_AddKernel << <(len1 / TPB), TPB >> >(d_inp1, len1, d_inp2, len2, d_out);

		hipMemcpy(out, d_out, len1*sizeof(float), hipMemcpyDeviceToHost);
		hipFree(d_inp1);
		hipFree(d_inp2);
		hipFree(d_out);
}

void component_Prod(float *inp1, int len1, float *inp2, int len2, float *out)
{
	
		float *d_inp1 = 0;
		float *d_inp2 = 0;
		float *d_out = 0;

		hipMalloc(&d_inp1, len1*sizeof(float));
		hipMalloc(&d_inp2, len1*sizeof(float));
		hipMalloc(&d_out, len1*sizeof(float));

		hipMemcpy(d_inp1, inp1, len1*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_inp2, inp2, len1*sizeof(float), hipMemcpyHostToDevice);

		component_ProdKernel << <(len1 / TPB), TPB >> >(d_inp1, len1, d_inp2, len2, d_out);

		hipMemcpy(out, d_out, len1*sizeof(float), hipMemcpyDeviceToHost);
		hipFree(d_inp1);
		hipFree(d_inp2);
		hipFree(d_out);
}


